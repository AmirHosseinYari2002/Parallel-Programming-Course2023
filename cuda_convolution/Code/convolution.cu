#include "hip/hip_runtime.h"
#include "convolution.h"

#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!


//-----------------------------------------------------------------------------

/*__global__ void kernelFunc(const float *f, const float *g, float *result, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n + n - 1 && col < n + n - 1) {
        float sum = 0.0f;

        for (int i = max(0, row - n + 1); i <= min(row, n - 1); ++i) {
            for (int j = max(0, col - n + 1); j <= min(col, n - 1); ++j) {
                int fRow = row - i;
                int fCol = col - j;
                sum += f[fRow * n + fCol] * g[i * n + j];
            }
        }

        result[row * (n + n - 1) + col] = sum;
    }
}*/


__global__ void kernelFunc(const float *f, const float *g, float *result, int n)
{
    // Define shared memory
    __shared__ float shared_f[32];
    __shared__ float shared_g[32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n + n - 1 && col < n + n - 1) {
        float sum = 0.0f;

        // Load indices of f and g to shared memory
        int fRowStart = max(0, row - n + 1);
        int fRowEnd = min(row, n - 1);
        int fColStart = max(0, col - n + 1);
        int fColEnd = min(col, n - 1);

        for (int i = fRowStart; i <= fRowEnd; ++i) {
            for (int j = fColStart; j <= fColEnd; ++j) {
                int fRow = row - i;
                int fCol = col - j;

                // Copy indices of f and g to shared memory
                shared_f[threadIdx.x] = f[fRow * n + fCol];
                shared_g[threadIdx.x] = g[i * n + j];

                sum += shared_f[threadIdx.x] * shared_g[threadIdx.x];
            }
        }

        result[row * (n + n - 1) + col] = sum;
    }
}
