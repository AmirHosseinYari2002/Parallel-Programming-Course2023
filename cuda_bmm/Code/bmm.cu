#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y

#define bx blockIdx.x
#define by blockIdx.y

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 32
#define TILEY 16

// you may define other parameters here!
// it's lower than tilex and tiley or bigger than both of them
#define TILE 128
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n / TILEX, n / TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX, TILEY);
	return dimBlock;
}

__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

	__shared__ float shared_ad[TILEY][TILE + 1]; 
	__shared__ float shared_bd[TILE + 1][TILEX];

	int global_row = by * TILEY + ty;
	int global_column = bx * TILEX + tx;
	float result = 0;

	for (int i = 0; i < n / TILE; ++i) {
		// Load tiles into shared memory with coalesced access for mad
		if (tx < TILE && ty < TILEY) {
			shared_ad[ty][tx] = ad[global_row * n + i * TILE + tx];
		}

		// Load tiles into shared memory with coalesced access for mbd
		if (tx < TILEX && ty < TILE) {
			shared_bd[ty][tx] = bd[(i * TILE + ty) * n + global_column];
		}

		__syncthreads();

		// Compute partial sum with coalesced access for both mad and mbd
		for (int k = 0; k < TILE; ++k) {
			result += shared_ad[ty][k] * shared_bd[k][tx];
		}

		__syncthreads();
	}
	cd[global_row * n + globalCol] = result;
}
