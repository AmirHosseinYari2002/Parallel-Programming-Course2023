#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "scan.h"
#include "hip/hip_runtime.h"
#include ""
#include "gpuerrors.h"
#include "gputimer.h"
#include <stdio.h>
#include <stdlib.h>
#include "helper.h"


#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z


#define bdx blockDim.x
#define bdy blockDim.y
#define bdz blockDim.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define gdx gridDim.x
#define gdy gridDim.y
#define gdz gridDim.z


// __global__ void kernelFunc( uint8_t* cb , const uint8_t* const ab , const uint8_t* const A , int i , const int n , uint8_t* alpha_to_g , uint8_t* index_of_g )
// {
//     int num = (by*gdx*bdx)+(bx*bdx)+(tx);
//     __shared__ uint8_t alpha_to_h[256];
//     __shared__ uint8_t index_of_h[256];
//     __shared__ uint8_t AB[16];
//     if( tx< 256)
//     {
//         alpha_to_h[tx] = alpha_to_g[tx];
//         index_of_h[tx] = index_of_g[tx];
//         if((tx) <16){
//             AB[tx] = A[tx];
//         }
//     }
//     __syncthreads();

//     if( num <  (4 * n) && ( num >= 4*(1<<(i)) )) 
//     {
//         int k = (num % 4);
//         int vector = (num - 4*(1<<(i))) - k;
//         uint8_t c[4];
//         for(int s = 0 ; s < 4 ; ++s)
//         {
//             c[s] = ab[(vector + s)];
//         }
//         uint8_t r = 0;
//         if( AB[k*4 + 0]!= 0 && c[0]!= 0){
//             r ^= alpha_to_h[(uint32_t(index_of_h[c[0]]) + uint32_t(index_of_h[AB[k * 4 + 0]]))%255];
//         }
//         if( AB[k*4 + 1]!= 0 && c[1]!= 0){
//             r ^= alpha_to_h[(uint32_t(index_of_h[c[1]]) + uint32_t(index_of_h[AB[k * 4 + 1]]))%255];
//         }
//         if( AB[k*4 + 2]!= 0 && c[2]!= 0){
//             r ^= alpha_to_h[(uint32_t(index_of_h[c[2]]) + uint32_t(index_of_h[AB[k * 4 + 2]]))%255];
//         }
//         if( AB[k*4 + 3]!= 0 && c[3]!= 0){
//             r ^= alpha_to_h[(uint32_t(index_of_h[c[3]]) + uint32_t(index_of_h[AB[k * 4 + 3]]))%255];
//         }
//         cb[num]  ^= r;
//     }
// }

__global__ void kernelFunc( uint8_t* cipherBlock , const uint8_t* const alphaBlock , const uint8_t* const inputArray , int iteration , const int arraySize , uint8_t* alphaToGlobal , uint8_t* indexOfGlobal )
{
    int globalIndex = (blockIdx.y*gridDim.x*blockDim.x)+(blockIdx.x*blockDim.x)+(threadIdx.x);
    __shared__ uint8_t alphaToShared[256];
    __shared__ uint8_t indexOfShared[256];
    __shared__ uint8_t inputBlock[16];
    if( threadIdx.x < 256)
    {
        alphaToShared[threadIdx.x] = alphaToGlobal[threadIdx.x];
        indexOfShared[threadIdx.x] = indexOfGlobal[threadIdx.x];
        if((threadIdx.x) <16){
            inputBlock[threadIdx.x] = inputArray[threadIdx.x];
        }
    }
    __syncthreads();

    if( globalIndex <  (4 * arraySize) && ( globalIndex >= 4*(1<<(iteration)) )) 
    {
        int remainder = (globalIndex % 4);
        int vectorIndex = (globalIndex - 4*(1<<(iteration))) - remainder;
        uint8_t cipherSubBlock[4];
        for(int s = 0 ; s < 4 ; ++s)
        {
            cipherSubBlock[s] = alphaBlock[(vectorIndex + s)];
        }
        uint8_t result = 0;
        if( inputBlock[remainder*4 + 0]!= 0 && cipherSubBlock[0]!= 0){
            result ^= alphaToShared[(uint32_t(indexOfShared[cipherSubBlock[0]]) + uint32_t(indexOfShared[inputBlock[remainder * 4 + 0]]))%255];
        }
        if( inputBlock[remainder*4 + 1]!= 0 && cipherSubBlock[1]!= 0){
            result ^= alphaToShared[(uint32_t(indexOfShared[cipherSubBlock[1]]) + uint32_t(indexOfShared[inputBlock[remainder * 4 + 1]]))%255];
        }
        if( inputBlock[remainder*4 + 2]!= 0 && cipherSubBlock[2]!= 0){
            result ^= alphaToShared[(uint32_t(indexOfShared[cipherSubBlock[2]]) + uint32_t(indexOfShared[inputBlock[remainder * 4 + 2]]))%255];
        }
        if( inputBlock[remainder*4 + 3]!= 0 && cipherSubBlock[3]!= 0){
            result ^= alphaToShared[(uint32_t(indexOfShared[cipherSubBlock[3]]) + uint32_t(indexOfShared[inputBlock[remainder * 4 + 3]]))%255];
        }
        cipherBlock[globalIndex]  ^= result;
    }
}



__global__ void kernelFunc2(uint8_t* inputArray, uint8_t* alphaLookup, uint8_t* indexLookup)
{
    __shared__ uint8_t sharedInputArray[4][4];

    sharedInputArray[threadIdx.y][threadIdx.x] = inputArray[threadIdx.y*4 + threadIdx.x];

    __syncthreads();

    uint8_t xorResult = 0;

    #pragma unroll
    for(int i = 0; i < 4; ++i)
    {
        uint8_t sharedValX = sharedInputArray[i][threadIdx.x];
        uint8_t sharedValY = sharedInputArray[threadIdx.y][i];
        xorResult ^= (sharedValX && sharedValY) ? alphaLookup[(uint32_t(indexLookup[sharedValX]) + uint32_t(indexLookup[sharedValY]))%255] : 0;
    }

    inputArray[threadIdx.y*4 + threadIdx.x] = xorResult;
}

__global__ void kernelFunc3(  uint8_t* inputArray , uint8_t* outputArray, int iteration , int arraySize)
{
	int globalIndex = (blockIdx.y * gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x) + threadIdx.x;
	if( globalIndex < (4 * arraySize) && globalIndex >= 4 * (1 << iteration))
	{
		outputArray[globalIndex] = inputArray[globalIndex];
	}
}


void gpuKernel(  const uint8_t* const a, const uint8_t* const matrix, uint8_t* c, const int m, const int n, uint8_t* alpha_to, uint8_t* index_of)
{
    uint8_t* ab;
 	uint8_t* cb;
	uint8_t* matrixb;
	uint8_t* alpha_to_g;
	uint8_t* index_of_g;
    HANDLE_ERROR(hipMalloc((void**)&ab , 4 * n * sizeof(uint8_t)));
    HANDLE_ERROR(hipMalloc((void**)&cb , 4 * n * sizeof(uint8_t)));
    HANDLE_ERROR(hipMalloc((void**)&matrixb , 4 * 4 * sizeof(uint8_t)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_to_g , 256 * sizeof(uint8_t)));
    HANDLE_ERROR(hipMalloc((void**)&index_of_g , 256 * sizeof(uint8_t)));
	HANDLE_ERROR(hipMemcpy(ab , a , 4 * n * (sizeof(uint8_t)) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(cb , a , 4 * n * (sizeof(uint8_t)) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_to_g , alpha_to , 256 * (sizeof(uint8_t)) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(index_of_g , index_of , 256 * (sizeof(uint8_t)) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(matrixb  , matrix , 4 * 4 * (sizeof(uint8_t)) , hipMemcpyHostToDevice));
	dim3 blockSize(1024);
    dim3 gridSize(512 , 512);
	dim3 block ( 4 , 4  );
	for(int i = 0 ; i < m ; ++i)
	{
		kernelFunc<<< gridSize , blockSize >>>(cb , ab , matrixb , i , n , alpha_to_g , index_of_g ) ;
		kernelFunc3<<< gridSize , blockSize >>>(cb , ab , i , n );
		kernelFunc2<<< 1 , block >>>( matrixb , alpha_to_g , index_of_g );
	}
	HANDLE_ERROR(hipMemcpy(c , cb , 4 * n * sizeof(uint8_t) , hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(ab));
	HANDLE_ERROR(hipFree(cb));
	HANDLE_ERROR(hipFree(matrixb));
	HANDLE_ERROR(hipFree(index_of_g));
	HANDLE_ERROR(hipFree(alpha_to_g));
 }
